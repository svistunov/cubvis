#include "hip/hip_runtime.h"
#raw
#define EPSILON 0.000001f
#end raw

texture<float, 1, hipReadModeElementType> v_tex;

texture<float, 1, hipReadModeElementType> n_tex;

__device__ __constant__ float design[$dN];

__device__ __constant__ int kernelN;

inline __host__ __device__ float3 operator*(float3 a, float b) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

inline __device__ float dot(float3 a, float3 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

inline __device__ float3 cross(float3 a, float3 b)
{ 
    return make_float3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x); 
}

inline __device__ float3 operator-(float3 a, float3 b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

inline __device__ float3 make_vert(float *V, int i) {
    return make_float3(V[i], V[i+1], V[i+2]);
}

inline __device__ float3 make_tex1dvert(texture<float, 1, hipReadModeElementType> t, int i) {
    return make_float3(tex1Dfetch(t, i), tex1Dfetch(t, i+1), tex1Dfetch(t, i+2));
}

inline __device__ float3 make_shift_point(float *V, int i, float3 normal, float val) {
    return make_float3(V[i]+normal.x*val, V[i+1]+normal.y*val, V[i+2]+normal.z*val);
}

inline __device__ float3 make_tex1dshift_point(texture<float, 1, hipReadModeElementType> t, int i, float3 normal, float val) {
    return make_float3(tex1Dfetch(t, i)+normal.x*val, tex1Dfetch(t, i+1)+normal.y*val, tex1Dfetch(t, i+2)+normal.z*val);
}

inline float rsqrtf(float x) {
    return 1.0f / sqrtf(x);
}



inline __device__ float3 normalize(float3 v) {
    float invLen = rsqrtf(dot(v, v));
    return make_float3(invLen * v.x, invLen * v.y, invLen * v.z);
}

//http://www.cs.virginia.edu/~gfx/Courses/2003/ImageSynthesis/papers/Acceleration/Fast%20MinimumStorage%20RayTriangle%20Intersection.pdf
__device__ int intr_tringle(float3 v0, float3 edge1, float3 edge2, float3 orig, float3 dir) {
    float3 tvec = orig - v0;
    float3 pvec = cross(dir, edge2);
    float  det  = dot(edge1, pvec);
    det = __fdividef(1.0f, det);

    float u = dot(tvec, pvec) * det;
    if (u < 0.0f || u > 1.0f)
        return 1;

    float3 qvec = cross(tvec, edge1);
    float v = dot(dir, qvec) * det;
    if (v < 0.0f || (u + v) > 1.0f)
        return 1;
        
    return 0;
}
